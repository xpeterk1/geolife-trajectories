#include "hip/hip_runtime.h"

#define __HIPCC__
#define __CUDA_ARCH__ 860
#define NUMTHREADS 256

#include "hip/hip_runtime.h"
#include "device_atomic_functions.h"
#include ""
#include "File.h"

__global__ void kernel(int* input) 
{
	input[0] = 123;
	__syncthreads();
}

void ahoj()
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	hipError_t cudaStatus = hipSetDevice(0);

	int pole[100];

	int* a;
	hipMalloc((void**)&a, sizeof(int));
	kernel <<< 1, 1 >>> (a);
	
	int outputval;
	cudaStatus = hipMemcpy(&outputval, a, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(a);
}