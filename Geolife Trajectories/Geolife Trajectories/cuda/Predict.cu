#include "hip/hip_runtime.h"
#include "Predict.h"

#define __HIPCC__
#define __CUDA_ARCH__ 860
#define NUMTHREADS 256

#include "hip/hip_runtime.h"
#include "device_atomic_functions.h"
#include ""

__global__ void kernel(glm::vec2* points, int n)
{
	points[0].x = n;
	__syncthreads();
}

std::vector<glm::vec2> compute_pois(std::vector<Trajectory> points)
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	hipError_t cudaStatus = hipSetDevice(0);

	glm::vec2* input_buffer = 0;
	int n = points[0].positions.size();
	int size_bytes = n * sizeof(glm::vec2);

	hipMalloc((void**)&input_buffer, size_bytes);
	hipMemcpy(input_buffer, & points[0].positions[0], size_bytes, hipMemcpyHostToDevice);

	kernel << < 1, 1 >> > (input_buffer, n);

	std::vector<glm::vec2> results;
	results.resize(n);
	hipMemcpy(&results[0], input_buffer, size_bytes, hipMemcpyDeviceToHost);
	
	hipFree(input_buffer);

	return std::vector<glm::vec2>();
}